
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "lib.cuh"

void scanArray(int *output, int *input, int size);
void checkCudaError(hipError_t err);

int main()
{
	// allocate arrays
    const int N = 5;
    int *in, *out;

	// managed memory can be accessed by host and gpu - it is slightly slower than hipMalloc + hipMemcpy
	hipMallocManaged(&in, N * sizeof(int));
	hipMallocManaged(&out, N * sizeof(int));

	for (int i = 0; i < N; i++) {
		in[i] = i + 1;
		out[i] = 0;
	}

	// launch the kernel
	naive_scan1<<<1, N, N>>>(out, in, N);

	checkCudaError(
		// check that the kernel was launched ok
		hipGetLastError()
	);
	checkCudaError(
		// check the kernel executed ok
		hipDeviceSynchronize()
	);

    printf("scan{1,2,3,4,5} = {%d,%d,%d,%d,%d}\n", out[0], out[1], out[2], out[3], out[4]);

	hipFree(in);
	hipFree(out);
    return 0;
}

void checkCudaError(hipError_t err) {
	if (err != hipSuccess) {
		fprintf(stderr, "cuda error: %s\n", hipGetErrorString(err));
		exit(0);
	}
}


// Helper function for using CUDA to add vectors in parallel.
//void scanArray(int *output, int *input, int size)
//{
//	int *device_output, *device_output;
//	hipError_t err;
//
//    // Allocate GPU buffers for three vectors (two input, one output)    .
//	err = hipMalloc((void**)&device_output, size * sizeof(int));
//	if (err != hipSuccess) {
//		fprintf(stderr, "error allocating memory: %s\n", hipGetErrorString(err));
//		exit(0);
//	}
//
//	err = hipMalloc((void**)&device_input, size * sizeof(int));
//	if (err != hipSuccess) {
//		fprintf(stderr, "error allocating memory: %s\n", hipGetErrorString(err));
//		exit(0);
//	}
//
//    // Copy input vectors from host memory to GPU buffers.
//    err = hipMemcpy(device_input, input, size * sizeof(int), hipMemcpyHostToDevice);
//	if (err != hipSuccess) {
//		fprintf(stderr, "error copying h to d: %s\n", hipGetErrorString(err));
//		exit(0);
//	}
//
//    // Launch a kernel on the GPU with one thread for each element.
//    naive_scan1<<<1, size>>>(device_output, device_input, size);
//
//    // Check for any errors launching the kernel
//    err = hipGetLastError();
//    if (err != hipSuccess) {
//        fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(err));
//		exit(0);
//    }
//    
//    // hipDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the execution.
//	err = hipDeviceSynchronize();
//	if (err != hipSuccess) {
//		fprintf(stderr, "sync failed: %s\n", hipGetErrorString(err));
//		exit(0);
//	}
//
//    // Copy output vector from GPU buffer to host memory.
//	err = hipMemcpy(output, device_output, size * sizeof(int), hipMemcpyDeviceToHost);
//	if (err != hipSuccess) {
//		fprintf(stderr, "copying d to h failed: %s\n", hipGetErrorString(err));
//		exit(0);
//	}
//
//    //hipFree(device_input);
//    //hipFree(device_output);
//}
