
#include <stdio.h>
#include <string.h>
#include "hip/hip_runtime.h"
#include ""

#include "lib.cuh"

void checkCudaError(char *message, hipError_t err);
void printArray(int* arr, int length, char* prefix);

int main()
{
	// allocate arrays
    int N = 128;
    int *in, *out;

	// managed memory can be accessed by host and gpu - it is slightly slower than hipMalloc + hipMemcpy
	hipMallocManaged(&in, N * sizeof(int));
	hipMallocManaged(&out, N * sizeof(int));

	// populate arrays
	for (int i = 0; i < N; i++) {
		in[i] = i + 1;
		out[i] = 0;
	}

	prescan<<<1, N/2, N*sizeof(int)>>>(out, in, N);

	checkCudaError(
		"kernel launch",
		hipGetLastError()
	);
	checkCudaError(
		"kernel execution",
		hipDeviceSynchronize()
	);


	printArray(in, N, "input array");
	printArray(out, N, "scanned array");

	hipFree(in);
	hipFree(out);
    return 0;
}

void checkCudaError(char *message, hipError_t err) {
	if (err != hipSuccess) {
		fprintf(stderr, message);
		fprintf(stderr, ": %s\n", hipGetErrorString(err));
		exit(0);
	}
}

void printArray(int* arr, int length, char* prefix) {
	char string[50] = "";
	strcat(strcat(string, prefix), ": {");

	printf(string);
	for (int i = 0; i < length; i++) {
		printf(" %i", arr[i]);
	}
	printf(" }\n");
}
