
#include <stdio.h>
#include <string.h>
#include "hip/hip_runtime.h"
#include ""

#include "lib.cuh"

void checkCudaError(char *message, hipError_t err);
void printArray(int* arr, int length, char* prefix);
bool isPowerOfTwo(int x);
int nextPowerOfTwo(int x);
void addConstant(int *output, int length, int constant);
void scanSmallArray(int *output, int *input, int length);
void scanLargeArray(int *output, int *input, int length);
void scanSmallArbitraryArray(int *output, int *input, int length);
void scanLargeArbitraryArray(int *output, int *input, int length);

int main()
{
	// allocate arrays
    const int N = 2046;

	int in[N];
	int out[N] = { 0 };

	// populate arrays
	for (int i = 0; i < N; i++) {
		in[i] = i + 1;
		out[i] = 0;
	}

	scanLargeArbitraryArray(out, in, N);

	printArray(in, N, "input array");
	printArray(out, N, "scanned array");

    return 0;
}

void scanSmallArray(int *output, int *input, int length) {
	const int arraySize = length * sizeof(int);
	int *d_out, *d_in;
	hipMalloc((void **)&d_out, arraySize);
	hipMalloc((void **)&d_in, arraySize);

	hipMemcpy(d_in, input, arraySize, hipMemcpyHostToDevice);

	prescan<<<1, length / 2, arraySize>>>(d_out, d_in, length);
	checkCudaError(
		"kernel launch",
		hipGetLastError()
	);
	checkCudaError(
		"kernel execution",
		hipDeviceSynchronize()
	);

	hipMemcpy(output, d_out, arraySize, hipMemcpyDeviceToHost);

	hipFree(d_out);
	hipFree(d_in);
}

void scanSmallArbitraryArray(int *output, int *input, int length) {
	const int arraySize = length * sizeof(int);
	int *d_out, *d_in;
	hipMalloc((void **)&d_out, arraySize);
	hipMalloc((void **)&d_in, arraySize);

	hipMemcpy(d_in, input, arraySize, hipMemcpyHostToDevice);

	int powerOfTwo = nextPowerOfTwo(length);

	prescan_arbitrary<<<1, length / 2, powerOfTwo * sizeof(int)>>>(d_out, d_in, length, powerOfTwo);
	checkCudaError(
		"kernel launch",
		hipGetLastError()
	);
	checkCudaError(
		"kernel execution",
		hipDeviceSynchronize()
	);

	hipMemcpy(output, d_out, arraySize, hipMemcpyDeviceToHost);

	hipFree(d_out);
	hipFree(d_in);
}

void scanLargeArray(int *output, int *input, int length) {
	const int numElementsPerBlock = 512;
	const int blocks = length / numElementsPerBlock;
	const int threadsPerBlock = numElementsPerBlock / 2;
	const int arraySize = length * sizeof(int);

	int *d_out, *d_in, *d_sums, *d_sums2, *d_incr;

	hipMalloc((void **)&d_out, arraySize);
	hipMalloc((void **)&d_in, arraySize);
	hipMalloc((void **)&d_sums, blocks * sizeof(int));
	hipMalloc((void **)&d_sums2, blocks * sizeof(int));
	hipMalloc((void **)&d_incr, blocks * sizeof(int));

	hipMemcpy(d_out, output, arraySize, hipMemcpyHostToDevice);
	hipMemcpy(d_in, input, arraySize, hipMemcpyHostToDevice);


	prescan_large<<<blocks, threadsPerBlock, arraySize>>>(d_out, d_in, numElementsPerBlock, d_sums);
	checkCudaError(
		"kernel launch",
		hipGetLastError()
	);
	checkCudaError(
		"kernel execution",
		hipDeviceSynchronize()
	);

	prescan<<<1, blocks / 2, blocks * sizeof(int)>>>(d_incr, d_sums, blocks);
	checkCudaError(
		"kernel launch",
		hipGetLastError()
	);
	checkCudaError(
		"kernel execution",
		hipDeviceSynchronize()
	);


	add<<<blocks, numElementsPerBlock>>>(d_out, d_incr, numElementsPerBlock);
	checkCudaError(
		"kernel launch",
		hipGetLastError()
	);
	checkCudaError(
		"kernel execution",
		hipDeviceSynchronize()
	);

	hipMemcpy(output, d_out, arraySize, hipMemcpyDeviceToHost);

	hipFree(d_out);
	hipFree(d_in);
	hipFree(d_sums);
}

void scanLargeArbitraryArray(int *output, int *input, int length) {
	bool isp2 = isPowerOfTwo(length);
	if (isp2) {
		scanLargeArray(output, input, length);
	}
	else {
		int prevPower = nextPowerOfTwo(length) / 2;
		int remaining = length - prevPower;

		scanLargeArray(output, input, prevPower);
		int last = output[prevPower - 1];

		scanSmallArbitraryArray(&(output[prevPower]), &(input[prevPower]), remaining);
		addConstant(&(output[prevPower]), remaining, last + input[prevPower - 1]);
	}
}


void addConstant(int *output, int length, int constant) {
	int *d_out, *d_add;
	int arraySize = length * sizeof(int);

	hipMalloc((void **)&d_out, arraySize);
	hipMalloc((void **)&d_add, sizeof(int));

	hipMemcpy(d_out, output, arraySize, hipMemcpyHostToDevice);
	hipMemcpy(d_add, &constant, sizeof(int), hipMemcpyHostToDevice);

	add << <1, length >> >(d_out, d_add, length);
	checkCudaError(
		"kernel launch",
		hipGetLastError()
	);
	checkCudaError(
		"kernel execution",
		hipDeviceSynchronize()
	);

	hipMemcpy(output, d_out, arraySize, hipMemcpyDeviceToHost);

	hipFree(d_out);
}

void checkCudaError(char *message, hipError_t err) {
	if (err != hipSuccess) {
		fprintf(stderr, message);
		fprintf(stderr, ": %s\n", hipGetErrorString(err));
		exit(0);
	}
}

void printArray(int* arr, int length, char* prefix) {
	char string[50] = "";
	strcat(strcat(string, prefix), ": {");

	printf(string);
	for (int i = 0; i < length; i++) {
		printf(" %i", arr[i]);
	}
	printf(" }\n");
}

bool isPowerOfTwo(int x) {
	return x && !(x & (x - 1));
}

int nextPowerOfTwo(int x) {
	int power = 1;
	while (power < x) {
		power *= 2;
	}
	return power;
}