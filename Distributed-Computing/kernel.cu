
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""

#include "lib.cuh"

void checkCudaError(hipError_t err);

int main()
{
	// allocate arrays
    const int N = 5;
    int *in, *out;

	// managed memory can be accessed by host and gpu - it is slightly slower than hipMalloc + hipMemcpy
	hipMallocManaged(&in, N * sizeof(int));
	hipMallocManaged(&out, N * sizeof(int));

	for (int i = 0; i < N; i++) {
		in[i] = i + 1;
		out[i] = 0;
	}

	// launch the kernel
	naive_scan<<<1, N, N>>>(out, in, N);

	checkCudaError(
		// check that the kernel was launched ok
		hipGetLastError()
	);
	checkCudaError(
		// check the kernel executed ok
		hipDeviceSynchronize()
	);

    printf("scan{1,2,3,4,5} = {%d,%d,%d,%d,%d}\n", out[0], out[1], out[2], out[3], out[4]);

	hipFree(in);
	hipFree(out);
    return 0;
}

void checkCudaError(hipError_t err) {
	if (err != hipSuccess) {
		fprintf(stderr, "cuda error: %s\n", hipGetErrorString(err));
		exit(0);
	}
}
