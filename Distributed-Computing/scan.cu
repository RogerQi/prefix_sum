#include "hip/hip_runtime.h"

#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""

#include "kernels.cuh"
#include "helpers.h"
#include "scan.cuh"

#define checkCudaError(o, l) _checkCudaError(o, l, __func__)

int THREADS_PER_BLOCK = 512;
int ELEMENTS_PER_BLOCK = THREADS_PER_BLOCK * 2;

long blockscan(int *output, int *input, int length, bool bcao){
	int *d_out, *d_in;
	const int arraySize = length * sizeof(int);

	hipMalloc((void **)&d_out, arraySize);
	hipMalloc((void **)&d_in, arraySize);
	hipMemcpy(d_out, output, arraySize, hipMemcpyHostToDevice);
	hipMemcpy(d_in, input, arraySize, hipMemcpyHostToDevice);

	// start timer
	long start = get_nanos();
	int powerOfTwo = nextPowerOfTwo(length);

	prescan_arbitrary<<<1, (length + 1) / 2, 2 * powerOfTwo * sizeof(int) >>>(d_out, d_in, length, powerOfTwo, bcao);
	checkCudaError(
		"kernel launch",
		hipGetLastError()
	);
	checkCudaError(
		"kernel execution",
		hipDeviceSynchronize()
	);

	// end timer
	hipDeviceSynchronize();
	long end = get_nanos();

	hipMemcpy(output, d_out, arraySize, hipMemcpyDeviceToHost);

	hipFree(d_out);
	hipFree(d_in);

	return end - start;
}


long scan(int *output, int *input, int length, bool bcao) {
	int *d_out, *d_in;
	const int arraySize = length * sizeof(int);

	hipMalloc((void **)&d_out, arraySize);
	hipMalloc((void **)&d_in, arraySize);
	hipMemcpy(d_out, output, arraySize, hipMemcpyHostToDevice);
	hipMemcpy(d_in, input, arraySize, hipMemcpyHostToDevice);

	// start timer
	long start = get_nanos();

	if (length > ELEMENTS_PER_BLOCK) {
		scanLargeDeviceArray(d_out, d_in, length, bcao);
	}
	else {
		scanSmallDeviceArray(d_out, d_in, length, bcao);
	}

	// end timer
	hipDeviceSynchronize();
	long end = get_nanos();

	hipMemcpy(output, d_out, arraySize, hipMemcpyDeviceToHost);

	hipFree(d_out);
	hipFree(d_in);

	return end - start;
}

void scanLargeDeviceArray(int *d_out, int *d_in, int length, bool bcao) {
	int remainder = length % (ELEMENTS_PER_BLOCK);
	if (remainder == 0) {
		scanLargeEvenDeviceArray(d_out, d_in, length, bcao);
	}
	else {
		// perform a large scan on a compatible multiple of elements
		int lengthMultiple = length - remainder;
		scanLargeEvenDeviceArray(d_out, d_in, lengthMultiple, bcao);

		// scan the remaining elements and add the last element of the large scan to this
		scanSmallDeviceArray(&(d_out[lengthMultiple]), &(d_in[lengthMultiple]), remainder, bcao);

		add<<<1, remainder>>>(&(d_out[lengthMultiple]), remainder, &(d_in[lengthMultiple - 1]), &(d_out[lengthMultiple - 1]));
		checkCudaError(
			"kernel launch",
			hipGetLastError()
		);
		checkCudaError(
			"kernel execution",
			hipDeviceSynchronize()
		);
	}
}

void scanSmallDeviceArray(int *d_out, int *d_in, int length, bool bcao) {
	int powerOfTwo = nextPowerOfTwo(length);

	prescan_arbitrary<<<1, (length + 1) / 2, 2 * powerOfTwo * sizeof(int)>>>(d_out, d_in, length, powerOfTwo, bcao);
	checkCudaError(
		"kernel launch",
		hipGetLastError()
	);
	checkCudaError(
		"kernel execution",
		hipDeviceSynchronize()
	);
}

void scanLargeEvenDeviceArray(int *d_out, int *d_in, int length, bool bcao) {
	const int blocks = length / ELEMENTS_PER_BLOCK;
	const int sharedMemArraySize = ELEMENTS_PER_BLOCK * sizeof(int);

	int *d_sums, *d_incr;
	hipMalloc((void **)&d_sums, blocks * sizeof(int));
	hipMalloc((void **)&d_incr, blocks * sizeof(int));

	prescan_large<<<blocks, THREADS_PER_BLOCK, 2 * sharedMemArraySize>>>(d_out, d_in, ELEMENTS_PER_BLOCK, d_sums, bcao);
	checkCudaError(
		"prescan_large kernel launch",
		hipGetLastError()
	);
	checkCudaError(
		"prescan_large kernel execution",
		hipDeviceSynchronize()
	);

	const int sumsArrThreadsNeeded = (blocks + 1) / 2;
	if (sumsArrThreadsNeeded > THREADS_PER_BLOCK) {
		// perform a large scan on the sums arr
		scanLargeDeviceArray(d_incr, d_sums, blocks, bcao);
	}
	else {
		// only need one block to scan sums arr so can use small scan
		scanSmallDeviceArray(d_incr, d_sums, blocks, bcao);
	}

	add<<<blocks, ELEMENTS_PER_BLOCK>>>(d_out, ELEMENTS_PER_BLOCK, d_incr);
	checkCudaError(
		"add kernel launch",
		hipGetLastError()
	);
	checkCudaError(
		"add kernel execution",
		hipDeviceSynchronize()
	);

	hipFree(d_sums);
	hipFree(d_incr);
}