#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>

#include "kernels.cuh"

#define SHARED_MEMORY_BANKS 32
#define LOG_MEM_BANKS 5

__device__ int CONFLICT_FREE_OFFSET(int n, bool bcao) {
	if (bcao) {
		return ((n) >> SHARED_MEMORY_BANKS + (n) >> (2 * LOG_MEM_BANKS));
	}
	else {
		return ((n) >> LOG_MEM_BANKS);
	}
}

__global__ void naive_scan(int *g_odata, int *g_idata, int n)
{
	extern __shared__ int temp[]; // allocated on invocation
	int k = threadIdx.x;
	// load input into shared memory.
	// This is exclusive scan, so shift right by one and set first elt to 0
	temp[k] = (k > 0) ? g_idata[k - 1] : 0;
	// sync threads so all elements in temp are allocated
	__syncthreads();

	for (int d = 1; d < n; d *= 2)
	{
		if (k >= d)
			temp[k] += temp[k - d];
		__syncthreads();
	}
	g_odata[k] = temp[k]; // write output
}

__global__ void prescan(int *output, int *input, int n, bool bcao) {
	extern __shared__ int temp[];

	int threadID = threadIdx.x;
	int offset = 1;

	int ai = threadID;
	int bi = threadID + (n / 2);
	int bankOffsetA = CONFLICT_FREE_OFFSET(ai, bcao);
	int bankOffsetB = CONFLICT_FREE_OFFSET(bi, bcao);
	temp[ai + bankOffsetA] = input[ai];
	temp[bi + bankOffsetB] = input[bi];

	for (int d = n >> 1; d > 0; d >>= 1) // build sum in place up the tree
	{
		__syncthreads();
		if (threadID < d)
		{
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			ai += CONFLICT_FREE_OFFSET(ai, bcao);
			bi += CONFLICT_FREE_OFFSET(bi, bcao);

			temp[bi] += temp[ai];
		}
		offset *= 2;
	}

	if (threadID == 0) { 
		temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1, bcao)] = 0; // clear the last element
	} 

	for (int d = 1; d < n; d *= 2) // traverse down tree & build scan
	{
		offset >>= 1;
		__syncthreads();
		if (threadID < d)
		{
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			ai += CONFLICT_FREE_OFFSET(ai, bcao);
			bi += CONFLICT_FREE_OFFSET(bi, bcao);

			int t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();

	output[ai] = temp[ai + bankOffsetA];
	output[bi] = temp[bi + bankOffsetB];
}

__global__ void prescan_arbitrary(int *output, int *input, int n, int powerOfTwo, bool bcao)
{
	extern __shared__ int temp[];// allocated on invocation
	int threadID = threadIdx.x;

	int ai = threadID;
	int bi = threadID + (n / 2);
	int bankOffsetA = CONFLICT_FREE_OFFSET(ai, bcao);
	int bankOffsetB = CONFLICT_FREE_OFFSET(bi, bcao);

	
	if (threadID < n) {
		temp[ai + bankOffsetA] = input[ai];
		temp[bi + bankOffsetB] = input[bi];
	}
	else {
		temp[ai + bankOffsetA] = 0;
		temp[bi + bankOffsetB] = 0;
	}
	

	int offset = 1;
	for (int d = powerOfTwo >> 1; d > 0; d >>= 1) // build sum in place up the tree
	{
		__syncthreads();
		if (threadID < d)
		{
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			ai += CONFLICT_FREE_OFFSET(ai, bcao);
			bi += CONFLICT_FREE_OFFSET(bi, bcao);

			temp[bi] += temp[ai];
		}
		offset *= 2;
	}

	if (threadID == 0) {
		temp[powerOfTwo - 1 + CONFLICT_FREE_OFFSET(powerOfTwo - 1, bcao)] = 0; // clear the last element
	}

	for (int d = 1; d < powerOfTwo; d *= 2) // traverse down tree & build scan
	{
		offset >>= 1;
		__syncthreads();
		if (threadID < d)
		{
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			ai += CONFLICT_FREE_OFFSET(ai, bcao);
			bi += CONFLICT_FREE_OFFSET(bi, bcao);

			int t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();

	if (threadID < n) {
		output[ai] = temp[ai + bankOffsetA];
		output[bi] = temp[bi + bankOffsetB];
	}
}

__global__ void prescan_large(int *output, int *input, int n, int *sums, bool bcao) {
	extern __shared__ int temp[];

	int blockID = blockIdx.x;
	int threadID = threadIdx.x;
	int blockOffset = blockID * n;
	
	int ai = threadID;
	int bi = threadID + (n / 2);
	int bankOffsetA = CONFLICT_FREE_OFFSET(ai, bcao);
	int bankOffsetB = CONFLICT_FREE_OFFSET(bi, bcao);
	temp[ai + bankOffsetA] = input[blockOffset + ai];
	temp[bi + bankOffsetB] = input[blockOffset + bi];

	int offset = 1;
	for (int d = n >> 1; d > 0; d >>= 1) // build sum in place up the tree
	{
		__syncthreads();
		if (threadID < d)
		{
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			ai += CONFLICT_FREE_OFFSET(ai, bcao);
			bi += CONFLICT_FREE_OFFSET(bi, bcao);

			temp[bi] += temp[ai];
		}
		offset *= 2;
	}
	__syncthreads();


	if (threadID == 0) { 
		sums[blockID] = temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1, bcao)];
		temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1, bcao)] = 0;
	} 
	
	for (int d = 1; d < n; d *= 2) // traverse down tree & build scan
	{
		offset >>= 1;
		__syncthreads();
		if (threadID < d)
		{
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			ai += CONFLICT_FREE_OFFSET(ai, bcao);
			bi += CONFLICT_FREE_OFFSET(bi, bcao);

			int t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();

	output[blockOffset + ai] = temp[ai + bankOffsetA];
	output[blockOffset + bi] = temp[bi + bankOffsetB];
}

__global__ void add(int *output, int length, int *n) {
	int blockID = blockIdx.x;
	int threadID = threadIdx.x;
	int blockOffset = blockID * length;

	output[blockOffset + threadID] += n[blockID];
}

__global__ void add(int *output, int length, int *n1, int *n2) {
	int blockID = blockIdx.x;
	int threadID = threadIdx.x;
	int blockOffset = blockID * length;

	output[blockOffset + threadID] += n1[blockID] + n2[blockID];
}